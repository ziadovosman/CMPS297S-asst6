#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

#define BLOCK_DIM 1024

__global__ void reduce_kernel(float* input, float* sum, unsigned int N) {

unsigned int segment = blockIdx.x*blockDim.x*2;
//to start at half, we add BLOCK_DIM
unsigned int i = segment + threadIdx.x + BLOCK_DIM;
//loop from 0 to BLOCK_DIM. meaning take from first element until half of the input
//since the threads start at the middle
for(unsigned int stride = 0; stride <= BLOCK_DIM; stride *= 2){
    if(threadIdx.x < stride){
        //i - BLOCK_DIM + stride
        input[i] += input[i - BLOCK_DIM + stride];
    }
    __syncthreads();
}


if(threadIdx.x == 0){
    sum[blockIdx.x] = input[i];
}









}

float reduce_gpu(float* input, unsigned int N) {

    Timer timer;

    // Allocate memory
    startTime(&timer);
    float *input_d;
    hipMalloc((void**) &input_d, N*sizeof(float));
    float *sum_d;
    hipMalloc((void**) &sum_d, sizeof(float));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);
    hipMemcpy(input_d, input, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemset(sum_d, 0, sizeof(float));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);
    const unsigned int numThreadsPerBlock = BLOCK_DIM;
    const unsigned int numElementsPerBlock = 2*numThreadsPerBlock;
    const unsigned int numBlocks = (N + numElementsPerBlock - 1)/numElementsPerBlock;
    reduce_kernel <<< numBlocks, numThreadsPerBlock >>> (input_d, sum_d, N);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);
    float sum;
    hipMemcpy(&sum, sum_d, sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free memory
    startTime(&timer);
    hipFree(input_d);
    hipFree(sum_d);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

    return sum;

}

